/*
!nvcc pi.cu -o pi -Wno-deprecated-gpu-targets -lcurand
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define NUM_BLOCKS 1024
#define THREADS_PER_BLOCK 256
#define ITER_PER_THREAD 2048

#define PI 3.14159265359

__global__ void kernel(int *count)
{
    double x, y, z;

    // find the overall ID of the thread
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    count[index] = 0;
    hiprandState state;
    hiprand_init((unsigned long long)clock() + index, 0, 0, &state);
    for (int i = 0; i < ITER_PER_THREAD; i++)
    {
        x = hiprand_uniform_double(&state);
        y = hiprand_uniform_double(&state);
        z =  x * x + y * y;
 
        if (z <= 1)
            count[index] += 1;
    }
}

void CUDAErrorCheck()
{
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
        exit(EXIT_FAILURE);
    }
}

int main()
{   
    long unsigned int n = NUM_BLOCKS * THREADS_PER_BLOCK;
    int *count;
    hipMallocManaged(&count, n * sizeof(int));
    CUDAErrorCheck();
        
    kernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(count);
        
    hipDeviceSynchronize();
    CUDAErrorCheck();
    
    long unsigned int reduced_count = 0;
    for(int i = 0; i < n; i++)
        reduced_count += count[i];
 
    hipFree(count);
 
    // find the ratio
    long unsigned int total_iter = n * ITER_PER_THREAD;
    double pi = ((double)reduced_count / total_iter) * 4.0;
    printf("PI [%lu iterations] = %.10g\n", total_iter, pi);
    printf("Error = %.10g\n", pi - PI);

    return 0;
}